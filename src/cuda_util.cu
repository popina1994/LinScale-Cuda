#include "cuda_util.h"
std::map<std::string, double> mapMemoryTrack;
std::map<std::string, double> mapMemoryStart;

const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch (status) {
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "Library not initialized";
        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "Resource allocation failed";
        case HIPBLAS_STATUS_INVALID_VALUE:
            return "Invalid value";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "Architecture mismatch";
        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "Memory mapping error";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "Execution failed";
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "Internal error";
        default:
            return "Unknown error";
    }
}

const char* cusolverGetErrorString(hipsolverStatus_t status)
{
    switch (status) {
        case HIPSOLVER_STATUS_SUCCESS: return "HIPSOLVER_STATUS_SUCCESS";
        case HIPSOLVER_STATUS_NOT_INITIALIZED: return "HIPSOLVER_STATUS_NOT_INITIALIZED";
        case HIPSOLVER_STATUS_ALLOC_FAILED: return "HIPSOLVER_STATUS_ALLOC_FAILED";
        case HIPSOLVER_STATUS_INVALID_VALUE: return "HIPSOLVER_STATUS_INVALID_VALUE";
        case HIPSOLVER_STATUS_ARCH_MISMATCH: return "HIPSOLVER_STATUS_ARCH_MISMATCH";
        case HIPSOLVER_STATUS_MAPPING_ERROR: return "HIPSOLVER_STATUS_MAPPING_ERROR";
        case HIPSOLVER_STATUS_EXECUTION_FAILED: return "HIPSOLVER_STATUS_EXECUTION_FAILED";
        case HIPSOLVER_STATUS_INTERNAL_ERROR: return "HIPSOLVER_STATUS_INTERNAL_ERROR";
        case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED: return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
        case HIPSOLVER_STATUS_NOT_SUPPORTED: return "HIPSOLVER_STATUS_NOT_SUPPORTED";
        case HIPSOLVER_STATUS_ZERO_PIVOT: return "HIPSOLVER_STATUS_ZERO_PIVOT";
        case CUSOLVER_STATUS_INVALID_LICENSE: return "CUSOLVER_STATUS_INVALID_LICENSE";
        default: return "Unknown cuSOLVER error";
    }
}

double getCudaMemoryUsage(void)
{
    size_t freeBytes;
    size_t totalBytes;

    hipError_t status = hipMemGetInfo(&freeBytes, &totalBytes);

    auto freeMB = freeBytes / (1024.0 * 1024);
    auto totalMB = totalBytes / (1024.0 * 1024);

    return totalMB - freeMB;
}

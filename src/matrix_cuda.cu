#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <random>
#include <iomanip>
#include "types.h"
#include "matrix.h"

// CUDA error check macro
#define CUDA_CALL(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __LINE__ << std::endl; \
            return EXIT_FAILURE; \
        } \
    } while (0)


// cuSOLVER error check macro
#define CUSOLVER_CALL(call) \
    do { \
        hipsolverStatus_t err = call; \
        if (err != HIPSOLVER_STATUS_SUCCESS) { \
            std::cerr << "cuSOLVER Error: " << err << " at " << __LINE__ << std::endl; \
            return EXIT_FAILURE; \
        } \
    } while (0)


__device__ double sqrt(double x);

template<typename T>
__global__ void computeHeadsAndTails(T* d_mat, int numRows, int numCols) {
    __shared__ T dataHeads  [1024];
    int colIdx = threadIdx.x;
    int headRowIdx = 0;

    if (colIdx < numCols)
    {
        dataHeads[colIdx] = d_mat[IDX_R(headRowIdx, colIdx, numRows, numCols)];
    }
    __syncthreads();
    for (int rowIdx = headRowIdx + 1; rowIdx < numRows; rowIdx++)
    {
        T i = rowIdx - headRowIdx + 1;
        if (colIdx < numCols)
        {
            T prevRowSum;
            T tailVal;
            prevRowSum = dataHeads[colIdx];
            T matVal = d_mat[IDX_R(rowIdx, colIdx, numRows, numCols)];
            dataHeads[colIdx] += matVal;
            tailVal = (matVal * (i - 1) - prevRowSum) / sqrt(i * (i - 1));
            d_mat[IDX_R(rowIdx, colIdx, numRows, numCols)] = tailVal;
            // printf("TAIL VAL %d %d %.3f %.3f\n", rowIdx, colIdx, i, tailVal);
        }
        __syncthreads();
    }
    if (colIdx < numCols)
    {
        d_mat[IDX_R(headRowIdx, colIdx, numRows, numCols)] = dataHeads[colIdx] / sqrt((double)numRows);
        // printf("HT: %.3f\n", dataHeads[colIdx] / sqrt(numRows));
    }
}

template <typename T>
__global__ void concatenateHeadsAndTails(const T* d_mat, const T* d_mat2Mod, T* dOutMat, int numRows1, int numCols1, int numRows2, int numCols2) {
    int colIdx = threadIdx.x;
    int headRowIdx = 0;
    const int numRowsOut = numRows1 + numRows2 - 1;
    const int numColsOut = numCols1 + numCols2;

    for (int rowIdx = 0; rowIdx < numRows1; rowIdx++)
    {
        if (colIdx < numCols1)
        {
            int posIdx = IDX_R(rowIdx, colIdx, numRowsOut, numColsOut);
            dOutMat[posIdx] = d_mat[IDX_R(rowIdx, colIdx, numRows1, numCols1)] * sqrt((double)numRows2);
            // printf("HERE 1 %d %d %.3f %d\n", rowIdx, colIdx, dOutMat[posIdx], posIdx);
        }
        if (colIdx < numCols2)
        {
            int posIdx2 = IDX_R(rowIdx, colIdx + numCols1, numRowsOut, numColsOut);
            dOutMat[posIdx2] = d_mat2Mod[IDX_R(headRowIdx, colIdx, numRows2, numCols2)];
            // printf("HERE 1 %d %d %.3f %d\n", rowIdx, colIdx + numCols1, dOutMat[posIdx2], posIdx2);
        }
    }
    for (int rowIdx = numRows1; rowIdx < numRowsOut; rowIdx++)
    {
        if (colIdx < numCols1)
        {
            int posIdx = IDX_R(rowIdx, colIdx, numRowsOut, numColsOut);
            dOutMat[posIdx] = 0;
            // printf("HERE 2 %d %d %.3f %d \n", rowIdx, colIdx, dOutMat[posIdx], posIdx);
        }
        if (colIdx < numCols2)
        {
            int posIdx2 = IDX_R(rowIdx, colIdx + numCols1, numRowsOut, numColsOut);
            dOutMat[posIdx2] = d_mat2Mod[IDX_R(rowIdx - numRows1 + 1, colIdx, numRows2, numCols2)] * sqrt((double)numRows1);
            // printf("HERE 2 %d %d %.3f %d\n", rowIdx, colIdx + numCols1, dOutMat[posIdx2], posIdx2);
        }
    }
}

template <typename T>
__global__ void setZerosUpperTriangular(T* d_A, int numRows, int numCols)
{
	int colIdx = threadIdx.x;
	for (int rowIdx = 0; rowIdx < numRows; rowIdx++)
	{
		if (rowIdx > colIdx)
		{
			d_A[IDX_C(rowIdx, colIdx, numRows, numCols)] = 0;
		}
	}
}

template <typename T>
int computeFigaro(const MatrixDRow& mat1, const MatrixDRow& mat2,
    Matrix<T, MajorOrder::COL_MAJOR>& matR, const std::string& fileName, int compute)
{
    int numRows1 = mat1.getNumRows();
    int numCols1 = mat1.getNumCols();
    int numRows2 = mat2.getNumRows();
    int numCols2 = mat2.getNumCols();
    int numRowsOut = numRows1 + numRows2 - 1;
    int numColsOut = numCols1 + numCols2;

    thrust::device_vector<T> d_mat1DV(mat1.getDataC(), mat1.getDataC() + numRows1 * numCols1);
    thrust::device_vector<T> d_mat2DV(mat2.getDataC(), mat2.getDataC() + numRows2 * numCols2);
    thrust::device_vector<T> d_matOutDV(numRowsOut * numColsOut);
    thrust::device_vector<T> d_matTranDV(numRowsOut * numColsOut);

    T* d_mat1 = thrust::raw_pointer_cast(d_mat1DV.data());
    T *d_mat2 = thrust::raw_pointer_cast(d_mat2DV.data());
    T* d_matOut = thrust::raw_pointer_cast(d_matOutDV.data());
    T *d_S;
    T* d_matOutTran = thrust::raw_pointer_cast(d_matTranDV.data());
    bool computeSVD = compute == 2;
    hipsolverHandle_t cusolverH;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));

    // Compute buffer size for QR
    int workspace_size = 0;
    if constexpr (std::is_same<T, float>::value)
    {
        CUSOLVER_CALL(hipsolverDnSgeqrf_bufferSize(cusolverH, numRowsOut, numColsOut, d_matOut, numRowsOut, &workspace_size));
    }
    else
    {
        CUSOLVER_CALL(hipsolverDnDgeqrf_bufferSize(cusolverH, numRowsOut, numColsOut, d_matOut, numRowsOut, &workspace_size));
    }

    // Initialize cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Allocate workspace
    T *d_work, *d_tau;
    CUDA_CALL(hipMalloc((void**)&d_work, workspace_size * sizeof(T)));

    // Allocate device status variable
    int *devInfo;
    CUDA_CALL(hipMalloc((void**)&devInfo, sizeof(int)));
    CUDA_CALL(hipMalloc((void**)&d_tau, std::min(numRowsOut, numColsOut) * sizeof(T)));

    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    // Start measuring time
    CUDA_CALL(hipEventRecord(start));

    // Compute join offsets for both tables
    // compute join offsets
    // for loop call for each subset the
    computeHeadsAndTails<<<1, numCols2>>>(d_mat2, numRows2, numCols2);
    concatenateHeadsAndTails<<<1, max(numCols1, numCols2)>>>(d_mat1, d_mat2, d_matOut, numRows1, numCols1, numRows2, numCols2);

    // Define scalars alpha and beta
    const T alpha = 1.0f; // Scalar for matrix A (no scaling)
    const T beta = 0.0f;  // Scalar for matrix B (no B matrix, so no scaling)

    if constexpr (std::is_same<T, float>::value)
    {
        hipblasSgeam(handle,
        HIPBLAS_OP_T, HIPBLAS_OP_T, // Transpose A (HIPBLAS_OP_T), no transpose for B (HIPBLAS_OP_N)
        numRowsOut, numColsOut,                     // Matrix dimensions
        &alpha,                   // Scalar for A
        d_matOut, numColsOut,                   // Input matrix A and its leading dimension
        &beta,                    // Scalar for B (not used)
        nullptr, numColsOut,               // No B matrix (set to nullptr)
        d_matOutTran, numRowsOut);                  // Output matrix C and its leading dimension
    }
    else
    {
        hipblasDgeam(handle,
        HIPBLAS_OP_T, HIPBLAS_OP_T, // Transpose A (HIPBLAS_OP_T), no transpose for B (HIPBLAS_OP_N)
        numRowsOut, numColsOut,                     // Matrix dimensions
        &alpha,                   // Scalar for A
        d_matOut, numColsOut,                   // Input matrix A and its leading dimension
        &beta,                    // Scalar for B (not used)
        nullptr, numColsOut,               // No B matrix (set to nullptr)
        d_matOutTran, numRowsOut);                  // Output matrix C and its leading dimension
    }

    int rank = min(numRowsOut, numColsOut);

    // Compute QR factorization
    if constexpr (std::is_same<T, float>::value)
    {
        CUSOLVER_CALL(hipsolverDnSgeqrf(cusolverH, numRowsOut, numColsOut, d_matOutTran, numRowsOut, d_tau, d_work, workspace_size, devInfo));
    }
    else
    {
        CUSOLVER_CALL(hipsolverDnDgeqrf (cusolverH, numRowsOut, numColsOut, d_matOutTran, numRowsOut, d_tau, d_work, workspace_size, devInfo));
        setZerosUpperTriangular<<<1, numColsOut>>>(d_matOutTran, numRowsOut, numColsOut);
    	if (computeSVD)
	    {
            std::cout << "WTF" << std::endl;
            char jobu = 'N';  // No computation of U
            char jobvt = 'N'; // No computation of V^T
            // cuSOLVER handle
            int *d_info;
            double *d_work;
            int lwork = 0;
            int ldA = numRowsOut;

            hipsolverHandle_t cusolverH1 = nullptr;
            CUSOLVER_CALL(hipsolverDnCreate(&cusolverH1));
            CUDA_CALL(hipMalloc((void**)&d_info, sizeof(int)));
            CUSOLVER_CALL(hipsolverDnDgesvd_bufferSize(cusolverH, rank, numColsOut, &lwork));
            CUDA_CALL(hipMalloc((void**)&d_work, sizeof(double) * lwork));
                CUDA_CALL(hipMalloc((void**)&d_S, sizeof(double) * rank));
            hipsolverDnDgesvd(cusolverH1, jobu, jobvt, numColsOut, numColsOut, d_matOutTran, ldA, d_S, nullptr, numColsOut, nullptr, numColsOut,
                            d_work, lwork, nullptr, d_info);
        }
    }

    // Stop measuring time
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));

    // Compute elapsed time
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));

    if (computeSVD)
    {
        thrust::host_vector<T> h_matS(numColsOut);
        T *h_S = thrust::raw_pointer_cast(h_matS.data());
        CUDA_CALL(hipMemcpy(h_S, d_S, numColsOut * sizeof(T), hipMemcpyDeviceToHost));
        printMatrix<T, MajorOrder::COL_MAJOR>(h_S, numColsOut, 1, numColsOut, fileName + "LinScaleS", false);
    }
    else
    {
    	thrust::host_vector<T> h_matOutH(numRowsOut * numColsOut);
    	T *h_matOut = thrust::raw_pointer_cast(h_matOutH.data());
    	CUDA_CALL(hipMemcpy(h_matOut, d_matOutTran, numRowsOut * numColsOut * sizeof(T), hipMemcpyDeviceToHost));
        matR = Matrix<T, MajorOrder::COL_MAJOR>{numColsOut, numColsOut};
        copyMatrix<T, MajorOrder::COL_MAJOR>(h_matOut, matR.getData(), numRowsOut, numColsOut, numColsOut, numColsOut, false);
    }


    CUDA_CALL(hipFree(d_tau));
    CUDA_CALL(hipFree(d_work));
    CUDA_CALL(hipFree(devInfo));
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));

    std::cout << "\n";
    if (computeSVD)
    {
	    std::cout << "SVD decomposition ";
    }
    else
    {
	    std::cout << "QR decomposition ";
    }
    std::cout << "Linscale took " << milliseconds << " ms.\n";

    return 0;
}

template <typename T, MajorOrder majorOrder>
int computeGeneral(const Matrix<T, majorOrder>& matA, Matrix<T, MajorOrder::COL_MAJOR>& matR, const std::string& fileName, int compute)
{
    // Allocate device memory
    T *d_A, *d_tau, *d_matOutTran, *h_S, *h_aCopy;
    int numRows = matA.getNumRows();
    int numCols = matA.getNumCols();

    thrust::device_vector<T> d_matA(matA.getDataC(), matA.getDataC() + numRows * numCols);
    thrust::host_vector<T> h_matACopy(numRows * numCols);
    thrust::device_vector<T> d_matADV(numRows * numCols);
    thrust::host_vector<T> h_matS(numCols);

    d_A = thrust::raw_pointer_cast(d_matA.data());
    d_matOutTran = thrust::raw_pointer_cast(d_matADV.data());
    h_S = thrust::raw_pointer_cast(h_matS.data());
    h_aCopy = thrust::raw_pointer_cast(h_matACopy.data());
    T *d_S;
    CUDA_CALL(hipMalloc((void**)&d_tau, std::min(numRows, numCols) * sizeof(T)));
    bool computeSVD = compute == 2;
     // Copy data to GPU
    if constexpr (majorOrder == MajorOrder::ROW_MAJOR)
    {
        // Initialize cuBLAS handle
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        // Define scalars alpha and beta
        const T alpha = 1.0f; // Scalar for matrix A (no scaling)
        const T beta = 0.0f;  // Scalar for matrix B (no B matrix, so no scaling)

        if constexpr (std::is_same<T, float>::value)
        {
            hipblasSgeam(handle,
            HIPBLAS_OP_T, HIPBLAS_OP_T, // Transpose A (HIPBLAS_OP_T), no transpose for B (HIPBLAS_OP_N)
            numRows, numCols,                     // Matrix dimensions
            &alpha,                   // Scalar for A
            d_A, numCols,                   // Input matrix A and its leading dimension
            &beta,                    // Scalar for B (not used)
            nullptr, numCols,               // No B matrix (set to nullptr)
            d_matOutTran, numRows);                  // Output matrix C and its leading dimension
        }
        else
        {
            hipblasDgeam(handle,
            HIPBLAS_OP_T, HIPBLAS_OP_T, // Transpose A (HIPBLAS_OP_T), no transpose for B (HIPBLAS_OP_N)
            numRows, numCols,                     // Matrix dimensions
            &alpha,                   // Scalar for A
            d_A, numCols,                   // Input matrix A and its leading dimension
            &beta,                    // Scalar for B (not used)
            nullptr, numCols,               // No B matrix (set to nullptr)
            d_matOutTran, numRows);                  // Output matrix C and its leading dimension
        }
        hipblasDestroy(handle);
    }
    else
    {
        d_matOutTran = d_A;
    }

    // cuSOLVER handle
    hipsolverHandle_t cusolverH;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));

    // Compute buffer size for QR
    int workspace_size = 0;
    if constexpr (std::is_same<T, float>::value)
    {
        CUSOLVER_CALL(hipsolverDnSgeqrf_bufferSize(cusolverH, numRows, numCols, d_matOutTran, numRows, &workspace_size));
    }
    else
    {
        CUSOLVER_CALL(hipsolverDnDgeqrf_bufferSize(cusolverH, numRows, numCols, d_matOutTran, numRows, &workspace_size));
    }
    // Allocate workspace
    T *d_work;
    CUDA_CALL(hipMalloc((void**)&d_work, workspace_size * sizeof(T)));

    // Allocate device status variable
    int *devInfo;
    CUDA_CALL(hipMalloc((void**)&devInfo, sizeof(int)));

    // CUDA event timing variables
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    // Start measuring time
    CUDA_CALL(hipEventRecord(start));

    // Compute QR factorization
    if constexpr (std::is_same<T, float>::value)
    {
        CUSOLVER_CALL(hipsolverDnSgeqrf(cusolverH, numRows, numCols, d_matOutTran, numRows, d_tau, d_work, workspace_size, devInfo));
    }
    else
    {
        if (computeSVD)
        {
            char jobu = 'N';  // No computation of U
            char jobvt = 'N'; // No computation of V^T
            // cuSOLVER handle
            int *d_info;
            double *d_work;
            int lwork = 0;
            int ldA = numRows;

            hipsolverHandle_t cusolverH1 = nullptr;
            CUSOLVER_CALL(hipsolverDnCreate(&cusolverH1));
            CUDA_CALL(hipMalloc((void**)&d_info, sizeof(int)));
            CUSOLVER_CALL(hipsolverDnDgesvd_bufferSize(cusolverH, numRows, numCols, &lwork));
            CUDA_CALL(hipMalloc((void**)&d_work, sizeof(double) * lwork));
            CUDA_CALL(hipMalloc((void**)&d_S, sizeof(double) * numCols));

            hipsolverDnDgesvd(cusolverH1, jobu, jobvt, numRows, numCols, d_matOutTran, ldA, d_S, nullptr, numRows, nullptr, numCols,
                                    d_work, lwork, nullptr, d_info);
        }
        else
        {
            CUSOLVER_CALL(hipsolverDnDgeqrf(cusolverH, numRows, numCols, d_matOutTran, numRows, d_tau, d_work, workspace_size, devInfo));
        }
    }

    // Stop measuring time
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));

    // Compute elapsed time
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));

    // Copy results back to host
    if (computeSVD)
    {
        CUDA_CALL(hipMemcpy(h_S, d_S, numCols * sizeof(T), hipMemcpyDeviceToHost));
        printMatrix<T, MajorOrder::COL_MAJOR>(h_S, numCols, 1, numCols, fileName + "cuSolverS", false);
    }
    else
    {
        CUDA_CALL(hipMemcpy(h_aCopy, d_matOutTran, numRows * numCols * sizeof(T), hipMemcpyDeviceToHost));
        matR = Matrix<T, MajorOrder::COL_MAJOR>{numCols, numCols};
        copyMatrix<T, MajorOrder::COL_MAJOR>(h_aCopy, matR.getData(), numRows, numCols, numCols, numCols, true);
    }

    // Print execution time
    std::string nameDecomp = computeSVD ? "SVD" : "QR";
    std::cout << "\n" + nameDecomp + " decomposition CUSolver took " << milliseconds << " ms.\n";

    // Cleanup
    CUDA_CALL(hipFree(d_tau));
    CUDA_CALL(hipFree(d_work));
    CUDA_CALL(hipFree(devInfo));
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));

    return 0;
}

template int computeGeneral<double, MajorOrder::ROW_MAJOR>(const MatrixDRow& matA,
    MatrixDCol& matR, const std::string& fileName, int compute);

template int computeGeneral<double, MajorOrder::COL_MAJOR>(const MatrixDCol& matA,
        MatrixDCol& matR, const std::string& fileName, int compute);

template int computeFigaro<double>(const MatrixDRow& mat1, const MatrixDRow& mat2,
    Matrix<double, MajorOrder::COL_MAJOR>& matR, const std::string& fileName, int compute);
#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <random>
#include <boost/program_options.hpp>

namespace po = boost::program_options;

// CUDA error check macro
#define CUDA_CALL(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __LINE__ << std::endl; \
            return EXIT_FAILURE; \
        } \
    } while (0)


// cuSOLVER error check macro
#define CUSOLVER_CALL(call) \
    do { \
        hipsolverStatus_t err = call; \
        if (err != HIPSOLVER_STATUS_SUCCESS) { \
            std::cerr << "cuSOLVER Error: " << err << " at " << __LINE__ << std::endl; \
            return EXIT_FAILURE; \
        } \
    } while (0)

enum class MajorOrder
{
    ROW_MAJOR = 0,
    COL_MAJOR = 1
};

#define IDX(rowIdx, colIdx, width) ((rowIdx) * (width) + (colIdx))
#define IDX_R(rowIdx, colIdx, numRows, numCols) ((rowIdx) * (numCols) + (colIdx) )
#define IDX_C(rowIdx, colIdx, numRows, numCols) ((rowIdx)  + (colIdx) * (numRows))

template <typename T, MajorOrder order>
void printMatrix(T* pArr, int numRows, int numCols, int numRowsCut, const std::string& fileName, bool upperTriangular = false)
{
    std::ofstream outFile(fileName);
    if (!outFile.is_open())
    {
        std::cerr << "WTF?" << fileName << std::endl;
    }
    for (int rowIdx = 0; rowIdx < min(numRows, numRowsCut); rowIdx++)
    {
        for (int colIdx = 0; colIdx < numCols; colIdx++)
        {
            if (upperTriangular and (rowIdx > colIdx))
            {
                outFile << "0";
            }
            else
            {
                if constexpr (order == MajorOrder::ROW_MAJOR)
                {
                    outFile << pArr[IDX_R(rowIdx, colIdx, numRows, numCols)];
                }
                else
                {
                    outFile << pArr[IDX_C(rowIdx, colIdx, numRows, numCols)];
                }
            }
            outFile << " ";
        }
        outFile << std::endl;
    }	
}

// column major version
template <typename T>
void generateRandom(T*& pArr, int numRows, int numCols, int offset)
{
    std::mt19937 gen(offset); // Fixed seed
    std::uniform_real_distribution<double> dist(0.0, 1.0);
    pArr = new T [numRows * numCols];
    // col_major
    for (int colIdx = 0; colIdx < numCols; colIdx++)
    {
        for (int rowIdx = 0; rowIdx < numRows; rowIdx++)
        {
            int pos = IDX_R(rowIdx, colIdx, numRows, numCols);
            pArr[pos] = dist(gen);
        }
    }
}

template<typename T, MajorOrder orderOutput>
void generateCartesianProduct(T* pArr1, T* pArr2, int numRows1, int numCols1, int numRows2, int numCols2, T*& pArr)
{
    int numRows = numRows1 * numRows2;
    int numCols =  numCols1 + numCols2;
    pArr = new T[numRows * numCols];
    for (int rowIdx = 0; rowIdx < numRows1 * numRows2; rowIdx++)
    {
        int rowIdx1 = rowIdx / numRows2;
        int rowIdx2 = rowIdx % numRows2;
        for (int colIdx = 0; colIdx < numCols1; colIdx++)
        {
            int pos;
            if constexpr (orderOutput == MajorOrder::ROW_MAJOR)
            {
                pos = IDX_R(rowIdx, colIdx, numRows, numCols);
            }
            else
            {
                pos = IDX_C(rowIdx, colIdx, numRows, numCols);
            }
            pArr[pos] =  pArr1[IDX_R(rowIdx1, colIdx, numRows1, numCols1)];
        }
        for (int colIdx = numCols1; colIdx < numCols; colIdx++)
        {
            int pos;
            if constexpr (orderOutput == MajorOrder::ROW_MAJOR)
            {
                pos = IDX_R(rowIdx, colIdx, numRows, numCols);
            }
            else
            {
                pos = IDX_C(rowIdx, colIdx, numRows, numCols);
            }
            pArr[pos] =  pArr2[IDX_R(rowIdx2, colIdx - numCols1, numRows2, numCols2)];
        }
    }
}

template<typename T>
__global__ void computeHeadsAndTails(T* d_mat, int numRows, int numCols) {
    __shared__ T dataHeads[1024];
    int colIdx = threadIdx.x;
    int headRowIdx = 0;

    if (colIdx < numCols)
    {
        dataHeads[colIdx] = d_mat[IDX_R(headRowIdx, colIdx, numRows, numCols)];
    }
    __syncthreads();
    for (int rowIdx = headRowIdx + 1; rowIdx < numRows; rowIdx++)
    {
        T i = rowIdx - headRowIdx + 1;
        if (colIdx < numCols)
        {
            T prevRowSum;
            T tailVal;
            prevRowSum = dataHeads[colIdx];
            T matVal = d_mat[IDX_R(rowIdx, colIdx, numRows, numCols)];
            dataHeads[colIdx] += matVal;
            tailVal = (matVal * (i - 1) - prevRowSum) / sqrtf(i * (i - 1));
            d_mat[IDX_R(rowIdx, colIdx, numRows, numCols)] = tailVal;
            // printf("TAIL VAL %d %d %.3f %.3f\n", rowIdx, colIdx, i, tailVal);
        }
        __syncthreads();
    }
    if (colIdx < numCols)
    {
        d_mat[IDX_R(headRowIdx, colIdx, numRows, numCols)] = dataHeads[colIdx] / sqrtf(numRows);
        // printf("HT: %.3f\n", dataHeads[colIdx] / sqrtf(numRows));
    }
}

template <typename T>
__global__ void concatenateHeadsAndTails(T* d_mat, T* d_mat2Mod, T* dOutMat, int numRows1, int numCols1, int numRows2, int numCols2) {
    int colIdx = threadIdx.x;
    int headRowIdx = 0;
    const int numRowsOut = numRows1 + numRows2 - 1;
    const int numColsOut = numCols1 + numCols2;

    for (int rowIdx = 0; rowIdx < numRows1; rowIdx++)
    {
        if (colIdx < numCols1)
        {
            int posIdx = IDX_R(rowIdx, colIdx, numRowsOut, numColsOut);
            dOutMat[posIdx] = d_mat[IDX_R(rowIdx, colIdx, numRows1, numCols1)] * sqrtf(numRows2);
            // printf("HERE 1 %d %d %.3f %d\n", rowIdx, colIdx, dOutMat[posIdx], posIdx);
        }
        if (colIdx < numCols2)
        {
            int posIdx2 = IDX_R(rowIdx, colIdx + numCols1, numRowsOut, numColsOut);
            dOutMat[posIdx2] = d_mat2Mod[IDX_R(headRowIdx, colIdx, numRows2, numCols2)];
            // printf("HERE 1 %d %d %.3f %d\n", rowIdx, colIdx + numCols1, dOutMat[posIdx2], posIdx2);
        }
    }
    for (int rowIdx = numRows1; rowIdx < numRowsOut; rowIdx++)
    {
        if (colIdx < numCols1)
        {
            int posIdx = IDX_R(rowIdx, colIdx, numRowsOut, numColsOut);
            dOutMat[posIdx] = 0;
            // printf("HERE 2 %d %d %.3f %d \n", rowIdx, colIdx, dOutMat[posIdx], posIdx);
        }
        if (colIdx < numCols2)
        {
            int posIdx2 = IDX_R(rowIdx, colIdx + numCols1, numRowsOut, numColsOut);
            dOutMat[posIdx2] = d_mat2Mod[IDX_R(rowIdx - numRows1 + 1, colIdx, numRows2, numCols2)] * sqrtf(numRows1);
            // printf("HERE 2 %d %d %.3f %d\n", rowIdx, colIdx + numCols1, dOutMat[posIdx2], posIdx2);
        }
    }
}

template <typename T> 
__global__ void setZerosUpperTriangular(T* d_A, int numRows, int numCols)
{
	int colIdx = threadIdx.x;
	for (int rowIdx = 0; rowIdx < numRows; rowIdx++)
	{
		if (rowIdx > colIdx)
		{
			d_A[IDX_C(rowIdx, colIdx, numRows, numCols)] = 0;
		}
	}
}

template <typename T>
int computeFigaro(T* h_mat1, T* h_mat2, int numRows1, int numCols1, int numRows2, int numCols2,
    std::string& fileName, int compute)
{
    int numRowsOut = numRows1 + numRows2 - 1;
    int numColsOut = numCols1 + numCols2;

    thrust::device_vector<T> d_mat1DV(h_mat1, h_mat1 + numRows1 * numCols1);
    thrust::device_vector<T> d_mat2DV(h_mat2, h_mat2 + numRows2 * numCols2);
    thrust::device_vector<T> d_matOutDV(numRowsOut * numColsOut);
    thrust::device_vector<T> d_matTranDV(numRowsOut * numColsOut);

    T* d_mat1 = thrust::raw_pointer_cast(d_mat1DV.data());
    T *d_mat2 = thrust::raw_pointer_cast(d_mat2DV.data());
    T* d_matOut = thrust::raw_pointer_cast(d_matOutDV.data());
    T *d_S;
    T* d_matOutTran = thrust::raw_pointer_cast(d_matTranDV.data());
    bool computeSVD = compute == 2;
    hipsolverHandle_t cusolverH;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));

    // Compute buffer size for QR
    int workspace_size = 0;
    if constexpr (std::is_same<T, float>::value)
    {
        CUSOLVER_CALL(hipsolverDnSgeqrf_bufferSize(cusolverH, numRowsOut, numColsOut, d_matOut, numRowsOut, &workspace_size));
    }
    else
    {
        CUSOLVER_CALL(hipsolverDnDgeqrf_bufferSize(cusolverH, numRowsOut, numColsOut, d_matOut, numRowsOut, &workspace_size));
    }

    // Initialize cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Allocate workspace
    T *d_work, *d_tau;
    CUDA_CALL(hipMalloc((void**)&d_work, workspace_size * sizeof(T)));

    // Allocate device status variable
    int *devInfo;
    CUDA_CALL(hipMalloc((void**)&devInfo, sizeof(int)));
    CUDA_CALL(hipMalloc((void**)&d_tau, std::min(numRowsOut, numColsOut) * sizeof(T)));

    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    // Start measuring time
    CUDA_CALL(hipEventRecord(start));

    // Compute join offsets for both tables
    // compute join offsets
    // for loop call for each subset the
    computeHeadsAndTails<<<1, numCols2>>>(d_mat2, numRows2, numCols2);
    concatenateHeadsAndTails<<<1, max(numCols1, numCols2)>>>(d_mat1, d_mat2, d_matOut, numRows1, numCols1, numRows2, numCols2);

    // Define scalars alpha and beta
    const T alpha = 1.0f; // Scalar for matrix A (no scaling)
    const T beta = 0.0f;  // Scalar for matrix B (no B matrix, so no scaling)

    if constexpr (std::is_same<T, float>::value)
    {
        hipblasSgeam(handle,
        HIPBLAS_OP_T, HIPBLAS_OP_T, // Transpose A (HIPBLAS_OP_T), no transpose for B (HIPBLAS_OP_N)
        numRowsOut, numColsOut,                     // Matrix dimensions
        &alpha,                   // Scalar for A
        d_matOut, numColsOut,                   // Input matrix A and its leading dimension
        &beta,                    // Scalar for B (not used)
        nullptr, numColsOut,               // No B matrix (set to nullptr)
        d_matOutTran, numRowsOut);                  // Output matrix C and its leading dimension
    }
    else
    {
        hipblasDgeam(handle,
        HIPBLAS_OP_T, HIPBLAS_OP_T, // Transpose A (HIPBLAS_OP_T), no transpose for B (HIPBLAS_OP_N)
        numRowsOut, numColsOut,                     // Matrix dimensions
        &alpha,                   // Scalar for A
        d_matOut, numColsOut,                   // Input matrix A and its leading dimension
        &beta,                    // Scalar for B (not used)
        nullptr, numColsOut,               // No B matrix (set to nullptr)
        d_matOutTran, numRowsOut);                  // Output matrix C and its leading dimension
    }
    int rank = min(numRowsOut, numColsOut);

    // // Compute QR factorization
    if constexpr (std::is_same<T, float>::value)
    {
        CUSOLVER_CALL(hipsolverDnSgeqrf(cusolverH, numRowsOut, numColsOut, d_matOutTran, numRowsOut, d_tau, d_work, workspace_size, devInfo));
    }
    else
    {
        CUSOLVER_CALL(hipsolverDnDgeqrf (cusolverH, numRowsOut, numColsOut, d_matOutTran, numRowsOut, d_tau, d_work, workspace_size, devInfo));
    	if (computeSVD)
	{
		setZerosUpperTriangular<<<1, numColsOut>>>(d_matOutTran, numRowsOut, numColsOut);
		char jobu = 'N';  // No computation of U
		char jobvt = 'N'; // No computation of V^T
		// cuSOLVER handle
		int *d_info;
		double *d_work;
		int lwork = 0;
		int ldA = numRowsOut;
	
		hipsolverHandle_t cusolverH1 = nullptr;
		CUSOLVER_CALL(hipsolverDnCreate(&cusolverH1));
		CUDA_CALL(hipMalloc((void**)&d_info, sizeof(int)));
		CUSOLVER_CALL(hipsolverDnDgesvd_bufferSize(cusolverH, rank, numColsOut, &lwork));
		CUDA_CALL(hipMalloc((void**)&d_work, sizeof(double) * lwork));
	    	CUDA_CALL(hipMalloc((void**)&d_S, sizeof(double) * rank));
		hipsolverDnDgesvd(cusolverH1, jobu, jobvt, numColsOut, numColsOut, d_matOutTran, ldA, d_S, nullptr, numColsOut, nullptr, numColsOut, 
				        d_work, lwork, nullptr, d_info);
        }	
    }

    // Stop measuring time
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));

    // Compute elapsed time
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
	
    if (computeSVD)
    {
	thrust::host_vector<T> h_matS(numColsOut);
    	T *h_S = thrust::raw_pointer_cast(h_matS.data());
	
	CUDA_CALL(hipMemcpy(h_S, d_S, numColsOut * sizeof(T), hipMemcpyDeviceToHost));
        printMatrix<T, MajorOrder::COL_MAJOR>(h_S, numColsOut, 1, numColsOut, fileName + "LinScaleS", false);
    }
    else 
    {
    	thrust::host_vector<T> h_matOutH(numRowsOut * numColsOut);
    	T *h_matOut = thrust::raw_pointer_cast(h_matOutH.data());
    	CUDA_CALL(hipMemcpy(h_matOut, d_matOutTran, numRowsOut * numColsOut * sizeof(T), hipMemcpyDeviceToHost));
    }
    //printMatrix<T, MajorOrder::COL_MAJOR>(h_matOut, numRowsOut, numColsOut, numColsOut, fileName + "LinScale", true);

    CUDA_CALL(hipFree(d_tau));
    CUDA_CALL(hipFree(d_work));
    CUDA_CALL(hipFree(devInfo));
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));

    delete [] h_mat1;
    delete [] h_mat2;
    std::cout << "\n";
    if (computeSVD)
    {
	    std::cout << "SVD decomposition ";
    }
    else 
    {
	    std::cout << "QR decomposition ";
    }
    std::cout << "Linscale took " << milliseconds << " ms.\n";

    return 0;
}

template <typename T, MajorOrder majorOrder>
int computeGeneral(T* h_A, int numRows, int numCols, const std::string& fileName, int compute)
{
    // Allocate device memory
    T *d_A, *d_tau, *d_matOutTran, *h_S;

    thrust::device_vector<T> d_matA(h_A, h_A + numRows * numCols);
    thrust::device_vector<T> d_matADV(numRows * numCols);
    thrust::host_vector<T> h_matS(numCols);
   
    d_A = thrust::raw_pointer_cast(d_matA.data());
    d_matOutTran = thrust::raw_pointer_cast(d_matADV.data());
    h_S = thrust::raw_pointer_cast(h_matS.data());
    T *d_S;
    CUDA_CALL(hipMalloc((void**)&d_tau, std::min(numRows, numCols) * sizeof(T)));
    bool computeSVD = compute == 2;
     // Copy data to GPU
    if constexpr (majorOrder == MajorOrder::ROW_MAJOR)
    {
        // Initialize cuBLAS handle
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        // Define scalars alpha and beta
        const T alpha = 1.0f; // Scalar for matrix A (no scaling)
        const T beta = 0.0f;  // Scalar for matrix B (no B matrix, so no scaling)

        if constexpr (std::is_same<T, float>::value)
        {
            hipblasSgeam(handle,
            HIPBLAS_OP_T, HIPBLAS_OP_T, // Transpose A (HIPBLAS_OP_T), no transpose for B (HIPBLAS_OP_N)
            numRows, numCols,                     // Matrix dimensions
            &alpha,                   // Scalar for A
            d_A, numCols,                   // Input matrix A and its leading dimension
            &beta,                    // Scalar for B (not used)
            nullptr, numCols,               // No B matrix (set to nullptr)
            d_matOutTran, numRows);                  // Output matrix C and its leading dimension
        }
        else
        {
            hipblasDgeam(handle,
            HIPBLAS_OP_T, HIPBLAS_OP_T, // Transpose A (HIPBLAS_OP_T), no transpose for B (HIPBLAS_OP_N)
            numRows, numCols,                     // Matrix dimensions
            &alpha,                   // Scalar for A
            d_A, numCols,                   // Input matrix A and its leading dimension
            &beta,                    // Scalar for B (not used)
            nullptr, numCols,               // No B matrix (set to nullptr)
            d_matOutTran, numRows);                  // Output matrix C and its leading dimension
        }
        hipblasDestroy(handle);
    }

    // cuSOLVER handle
    hipsolverHandle_t cusolverH;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));

    // Compute buffer size for QR
    int workspace_size = 0;
    if constexpr (std::is_same<T, float>::value)
    {
        CUSOLVER_CALL(hipsolverDnSgeqrf_bufferSize(cusolverH, numRows, numCols, d_matOutTran, numRows, &workspace_size));
    }
    else
    {
        CUSOLVER_CALL(hipsolverDnDgeqrf_bufferSize(cusolverH, numRows, numCols, d_matOutTran, numRows, &workspace_size));
    }
    // Allocate workspace
    T *d_work;
    CUDA_CALL(hipMalloc((void**)&d_work, workspace_size * sizeof(T)));

    // Allocate device status variable
    int *devInfo;
    CUDA_CALL(hipMalloc((void**)&devInfo, sizeof(int)));

    // CUDA event timing variables
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    // Start measuring time
    CUDA_CALL(hipEventRecord(start));

    // Compute QR factorization

    if constexpr (std::is_same<T, float>::value)
    {
        CUSOLVER_CALL(hipsolverDnSgeqrf(cusolverH, numRows, numCols, d_matOutTran, numRows, d_tau, d_work, workspace_size, devInfo));
    }
    else
    {
	if (computeSVD)
        {
                char jobu = 'N';  // No computation of U
                char jobvt = 'N'; // No computation of V^T
                // cuSOLVER handle
                int *d_info;
                double *d_work;
                int lwork = 0;
                int ldA = numRows;

                hipsolverHandle_t cusolverH1 = nullptr;
                CUSOLVER_CALL(hipsolverDnCreate(&cusolverH1));
                CUDA_CALL(hipMalloc((void**)&d_info, sizeof(int)));
                CUSOLVER_CALL(hipsolverDnDgesvd_bufferSize(cusolverH, numRows, numCols, &lwork));
                CUDA_CALL(hipMalloc((void**)&d_work, sizeof(double) * lwork));
		CUDA_CALL(hipMalloc((void**)&d_S, sizeof(double) * numCols));

                hipsolverDnDgesvd(cusolverH1, jobu, jobvt, numRows, numCols, d_matOutTran, ldA, d_S, nullptr, numRows, nullptr, numCols,
                                        d_work, lwork, nullptr, d_info);

        }
	else 
	{
        	CUSOLVER_CALL(hipsolverDnDgeqrf(cusolverH, numRows, numCols, d_matOutTran, numRows, d_tau, d_work, workspace_size, devInfo));
    
	}
    }

    // Stop measuring time
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));

    // Compute elapsed time
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));

    // Copy results back to host
    if (computeSVD)
    {
        CUDA_CALL(hipMemcpy(h_S, d_S, numCols * sizeof(T), hipMemcpyDeviceToHost));
        printMatrix<T, MajorOrder::COL_MAJOR>(h_S, numCols, 1, numCols, fileName + "cuSolverS", false);
    }
    else 
    {
        CUDA_CALL(hipMemcpy(h_A, d_matOutTran, numRows * numCols * sizeof(T), hipMemcpyDeviceToHost));
    }
    	 
    CUDA_CALL(hipMemcpy(h_A, d_matOutTran, numRows * numCols * sizeof(T), hipMemcpyDeviceToHost));

    //printMatrix<T, MajorOrder::COL_MAJOR>(h_A, numRows, numCols, numCols, fileName + "CUDA", true);

    // Print execution time
    std::string nameDecomp = computeSVD ? "SVD" : "QR"; 
    std::cout << "\n" + nameDecomp + " decomposition CUSolver took " << milliseconds << " ms.\n";

    // Cleanup
    CUDA_CALL(hipFree(d_tau));
    CUDA_CALL(hipFree(d_work));
    CUDA_CALL(hipFree(devInfo));
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));

    return 0;
}

void evaluate(int numRows1, int numCols1, int numRows2, int numCols2, std::string& fileName, int compute)
{
    double *h_mat1, *h_mat2, *pArr;
    generateRandom(h_mat1, numRows1, numCols1, 0);
    generateRandom(h_mat2, numRows2, numCols2, 10);
    // printMatrix<double, MajorOrder::ROW_MAJOR>(h_mat1, numRows, numCols, numRows, false);
    // printMatrix<double, MajorOrder::ROW_MAJOR>(h_mat2, numRows, numCols, numRows, false);
	
    generateCartesianProduct<double, MajorOrder::ROW_MAJOR>(h_mat1, h_mat2, numRows1, numCols1, numRows2, numCols2, pArr);
    //printMatrix<double, MajorOrder::ROW_MAJOR>(pArr, numRows1 * numRows2, numCols1 + numCols2, numRows1 * numRows2, "mat.csv", false);

    computeGeneral<double, MajorOrder::ROW_MAJOR>(pArr, numRows1 * numRows2, numCols1 + numCols2, fileName, compute);
    computeFigaro<double>(h_mat1, h_mat2, numRows1, numCols1, numRows2, numCols2, fileName, compute);
}

int main(int argc, char* argv[])
{
    int numRows1 = 1000, numCols1 = 4;
    int numRows2 = 2, numCols2 = 4;
    int compute = 1;
    try {
        // Define the command-line options
        po::options_description desc("Allowed options");
        desc.add_options()
            ("help,h", "Show help message")
            ("input,i", po::value<std::string>(), "Input file")
            ("m1", po::value<int>(), "Number of rows 1")
            ("m2", po::value<int>(), "Number of rows 2")
            ("n1", po::value<int>(), "Number of columns 1")
            ("n2", po::value<int>(), "Number of columns 2")
            ("compute", po::value<int>(), "Compute mode")
            ("verbose,v", "Enable verbose mode");

        // Parse the command-line arguments
        po::variables_map vm;
        po::store(po::parse_command_line(argc, argv, desc), vm);
        po::notify(vm);

        // Handle the help flag
        if (vm.count("help")) {
            std::cout << desc << std::endl;
            return 0;
        }
        if (vm.count("m1"))
        {
            numRows1 = vm["m1"].as<int>();
        }
        if (vm.count("m2"))
        {
            numRows2 = vm["m2"].as<int>();
        }
        if (vm.count("n1"))
        {
            numCols1 = vm["n1"].as<int>();
        }
        if (vm.count("n2"))
        {
            numCols2 = vm["n2"].as<int>();
        }
	if (vm.count("compute"))
	{
		compute = vm["compute"].as<int>();
	}
        std::string fileName = "results/" + std::to_string(numRows1) + "x" + std::to_string(numCols1) + "," + std::to_string(numRows2) + "x" + std::to_string(numCols2);
        evaluate(numRows1, numCols1, numRows2, numCols2, fileName, compute);
    } catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
    }

    return 0;
}

